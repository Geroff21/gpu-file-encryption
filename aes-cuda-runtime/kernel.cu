﻿#include "functions.h"

void printBytes(BYTE s[], int len) {
    for (int i = 0; i < len; i++)
        printf("%x ", s[i]);
    printf("\n");
}

void printChars(BYTE s[], int len) {
    for (int i = 0; i < len; i++)
        printf("%c", s[i]);
    printf("\n");
}

//Sbox
BYTE AES_Sbox_init[] =
{
    0x63,0x7c,0x77,0x7b,0xf2,0x6b,0x6f,0xc5,0x30,0x01,0x67,0x2b,0xfe,0xd7,0xab,0x76,
    0xca,0x82,0xc9,0x7d,0xfa,0x59,0x47,0xf0,0xad,0xd4,0xa2,0xaf,0x9c,0xa4,0x72,0xc0,
    0xb7,0xfd,0x93,0x26,0x36,0x3f,0xf7,0xcc,0x34,0xa5,0xe5,0xf1,0x71,0xd8,0x31,0x15,
    0x04,0xc7,0x23,0xc3,0x18,0x96,0x05,0x9a,0x07,0x12,0x80,0xe2,0xeb,0x27,0xb2,0x75,
    0x09,0x83,0x2c,0x1a,0x1b,0x6e,0x5a,0xa0,0x52,0x3b,0xd6,0xb3,0x29,0xe3,0x2f,0x84,
    0x53,0xd1,0x00,0xed,0x20,0xfc,0xb1,0x5b,0x6a,0xcb,0xbe,0x39,0x4a,0x4c,0x58,0xcf,
    0xd0,0xef,0xaa,0xfb,0x43,0x4d,0x33,0x85,0x45,0xf9,0x02,0x7f,0x50,0x3c,0x9f,0xa8,
    0x51,0xa3,0x40,0x8f,0x92,0x9d,0x38,0xf5,0xbc,0xb6,0xda,0x21,0x10,0xff,0xf3,0xd2,
    0xcd,0x0c,0x13,0xec,0x5f,0x97,0x44,0x17,0xc4,0xa7,0x7e,0x3d,0x64,0x5d,0x19,0x73,
    0x60,0x81,0x4f,0xdc,0x22,0x2a,0x90,0x88,0x46,0xee,0xb8,0x14,0xde,0x5e,0x0b,0xdb,
    0xe0,0x32,0x3a,0x0a,0x49,0x06,0x24,0x5c,0xc2,0xd3,0xac,0x62,0x91,0x95,0xe4,0x79,
    0xe7,0xc8,0x37,0x6d,0x8d,0xd5,0x4e,0xa9,0x6c,0x56,0xf4,0xea,0x65,0x7a,0xae,0x08,
    0xba,0x78,0x25,0x2e,0x1c,0xa6,0xb4,0xc6,0xe8,0xdd,0x74,0x1f,0x4b,0xbd,0x8b,0x8a,
    0x70,0x3e,0xb5,0x66,0x48,0x03,0xf6,0x0e,0x61,0x35,0x57,0xb9,0x86,0xc1,0x1d,0x9e,
    0xe1,0xf8,0x98,0x11,0x69,0xd9,0x8e,0x94,0x9b,0x1e,0x87,0xe9,0xce,0x55,0x28,0xdf,
    0x8c,0xa1,0x89,0x0d,0xbf,0xe6,0x42,0x68,0x41,0x99,0x2d,0x0f,0xb0,0x54,0xbb,0x16
};

//Sbox inverse
BYTE AES_Sbox_Inv_init[] =
{
    0x52,0x09,0x6a,0xd5,0x30,0x36,0xa5,0x38,0xbf,0x40,0xa3,0x9e,0x81,0xf3,0xd7,0xfb,
    0x7c,0xe3,0x39,0x82,0x9b,0x2f,0xff,0x87,0x34,0x8e,0x43,0x44,0xc4,0xde,0xe9,0xcb,
    0x54,0x7b,0x94,0x32,0xa6,0xc2,0x23,0x3d,0xee,0x4c,0x95,0x0b,0x42,0xfa,0xc3,0x4e,
    0x08,0x2e,0xa1,0x66,0x28,0xd9,0x24,0xb2,0x76,0x5b,0xa2,0x49,0x6d,0x8b,0xd1,0x25,
    0x72,0xf8,0xf6,0x64,0x86,0x68,0x98,0x16,0xd4,0xa4,0x5c,0xcc,0x5d,0x65,0xb6,0x92,
    0x6c,0x70,0x48,0x50,0xfd,0xed,0xb9,0xda,0x5e,0x15,0x46,0x57,0xa7,0x8d,0x9d,0x84,
    0x90,0xd8,0xab,0x00,0x8c,0xbc,0xd3,0x0a,0xf7,0xe4,0x58,0x05,0xb8,0xb3,0x45,0x06,
    0xd0,0x2c,0x1e,0x8f,0xca,0x3f,0x0f,0x02,0xc1,0xaf,0xbd,0x03,0x01,0x13,0x8a,0x6b,
    0x3a,0x91,0x11,0x41,0x4f,0x67,0xdc,0xea,0x97,0xf2,0xcf,0xce,0xf0,0xb4,0xe6,0x73,
    0x96,0xac,0x74,0x22,0xe7,0xad,0x35,0x85,0xe2,0xf9,0x37,0xe8,0x1c,0x75,0xdf,0x6e,
    0x47,0xf1,0x1a,0x71,0x1d,0x29,0xc5,0x89,0x6f,0xb7,0x62,0x0e,0xaa,0x18,0xbe,0x1b,
    0xfc,0x56,0x3e,0x4b,0xc6,0xd2,0x79,0x20,0x9a,0xdb,0xc0,0xfe,0x78,0xcd,0x5a,0xf4,
    0x1f,0xdd,0xa8,0x33,0x88,0x07,0xc7,0x31,0xb1,0x12,0x10,0x59,0x27,0x80,0xec,0x5f,
    0x60,0x51,0x7f,0xa9,0x19,0xb5,0x4a,0x0d,0x2d,0xe5,0x7a,0x9f,0x93,0xc9,0x9c,0xef,
    0xa0,0xe0,0x3b,0x4d,0xae,0x2a,0xf5,0xb0,0xc8,0xeb,0xbb,0x3c,0x83,0x53,0x99,0x61,
    0x17,0x2b,0x04,0x7e,0xba,0x77,0xd6,0x26,0xe1,0x69,0x14,0x63,0x55,0x21,0x0c,0x7d
};

/* ----- AES FUNCTIONS ----- */

void AES_SubBytes(BYTE state[], BYTE sbox[]) {
    for (int i = 0; i < 16; i++)
        state[i] = sbox[state[i]];
}

void AES_AddRoundKey(BYTE state[], BYTE rkey[]) {
    for (int i = 0; i < 16; i++)
        state[i] ^= rkey[i];
}

void AES_ShiftRows(BYTE state[], BYTE shifttab[]) {
    BYTE temp[16];
    for (int i = 0; i < 16; i++)
        temp[i] = state[shifttab[i]];
    memcpy(state, temp, 16);
}

void AES_MixColumns(BYTE state[], BYTE AES_xtime[]) {
    for (int i = 0; i < 16; i += 4) {
        BYTE s0 = state[i + 0], s1 = state[i + 1];
        BYTE s2 = state[i + 2], s3 = state[i + 3];
        BYTE h = s0 ^ s1 ^ s2 ^ s3;
        state[i + 0] ^= h ^ AES_xtime[s0 ^ s1];
        state[i + 1] ^= h ^ AES_xtime[s1 ^ s2];
        state[i + 2] ^= h ^ AES_xtime[s2 ^ s3];
        state[i + 3] ^= h ^ AES_xtime[s3 ^ s0];
    }
}

void AES_MixColumns_Inv(BYTE state[], BYTE AES_xtime[]) {
    for (int i = 0; i < 16; i += 4) {
        BYTE s0 = state[i + 0], s1 = state[i + 1];
        BYTE s2 = state[i + 2], s3 = state[i + 3];
        BYTE h = s0 ^ s1 ^ s2 ^ s3;
        BYTE xh = AES_xtime[h];
        BYTE h1 = AES_xtime[AES_xtime[xh ^ s0 ^ s2]] ^ h;
        BYTE h2 = AES_xtime[AES_xtime[xh ^ s1 ^ s3]] ^ h;
        state[i + 0] ^= h1 ^ AES_xtime[s0 ^ s1];
        state[i + 1] ^= h2 ^ AES_xtime[s1 ^ s2];
        state[i + 2] ^= h1 ^ AES_xtime[s2 ^ s3];
        state[i + 3] ^= h2 ^ AES_xtime[s3 ^ s0];
    }
}

int AES_ExpandKey(BYTE key[], int keyLen, BYTE AES_Sbox[]) {
    int kl = keyLen, ks, Rcon = 1, i, j;
    BYTE* temp;

    switch (kl) {
    case 16: ks = 16 * (10 + 1); break;
    case 24: ks = 16 * (12 + 1); break;
    case 32: ks = 16 * (14 + 1); break;
    default:
        fprintf(stderr, "Error: Only 16, 24, or 32 key lengths are allowed.\n");
        return -1;
    }

    for (i = kl; i < ks; i += 4) {
        temp = &key[i - 4];
        if (i % kl == 0) {
            BYTE tmp = temp[0];
            temp[0] = AES_Sbox[temp[1]] ^ Rcon;
            temp[1] = AES_Sbox[temp[2]];
            temp[2] = AES_Sbox[temp[3]];
            temp[3] = AES_Sbox[tmp];
            if ((Rcon <<= 1) >= 256)
                Rcon ^= 0x11b;
        }
        else if ((kl > 24) && (i % kl == 16)) {
            for (j = 0; j < 4; j++)
                temp[j] = AES_Sbox[temp[j]];
        }
        for (j = 0; j < 4; j++)
            key[i + j] = key[i + j - kl] ^ temp[j];
    }
    return ks;
}

// AES_Encrypt & AES_Decrypt

void AES_Initialize_Encrypt(BYTE AES_ShiftRowTab[], BYTE AES_xtime[]) {

    int i;

    for (i = 0; i < 16; ++i) {
        AES_ShiftRowTab[i] = (i % 4) * 4 + (i / 4);
    }

    for (i = 0; i < 128; i++) {
        AES_xtime[i] = i << 1;
        AES_xtime[128 + i] = (i << 1) ^ 0x1b;
    }
}

void AES_Initialize_Decrypt(BYTE AES_ShiftRowTab_Inv[], BYTE AES_xtime[]) {

    int i;

    for (i = 0; i < 16; i++)
        AES_ShiftRowTab_Inv[(i % 4) * 4 + (i / 4)] = i;

    for (i = 0; i < 128; i++) {
        AES_xtime[i] = i << 1;
        AES_xtime[128 + i] = (i << 1) ^ 0x1b;
    }
}

void AES_Encrypt(AES_block aes_block_array[], BYTE key[], int keyLen, int block_number) {

    BYTE AES_ShiftRowTab[16];
    BYTE AES_xtime[256];

    AES_Initialize_Encrypt(AES_ShiftRowTab, AES_xtime);

    BYTE block[16];

    for (int i = 0; i < 16; i++) {
        block[i] = aes_block_array[0].block[i];
    }

    int l = keyLen, i;

    AES_AddRoundKey(block, &key[0]);
    for (i = 16; i < l - 16; i += 16) {
        AES_SubBytes(block, AES_Sbox_init);
        AES_ShiftRows(block, AES_ShiftRowTab);
        AES_MixColumns(block, AES_xtime);
        AES_AddRoundKey(block, &key[i]);
    }
    AES_SubBytes(block, AES_Sbox_init);
    AES_ShiftRows(block, AES_ShiftRowTab);
    AES_AddRoundKey(block, &key[i]);

    for (int i = 0; i < 16; i++) {
        aes_block_array[0].block[i] = block[i];
    }

}

void AES_Decrypt(AES_block aes_block_array[], BYTE key[], int keyLen, int block_number) {

    BYTE AES_ShiftRowTab_Inv[16];
    BYTE AES_xtime[256];

    AES_Initialize_Decrypt(AES_ShiftRowTab_Inv, AES_xtime);

    BYTE block[16];
    for (int i = 0; i < 16; i++) {
        block[i] = aes_block_array[0].block[i];
    }

    int l = keyLen, i;
    AES_AddRoundKey(block, &key[l - 16]);
    AES_ShiftRows(block, AES_ShiftRowTab_Inv);
    AES_SubBytes(block, AES_Sbox_Inv_init);
    for (i = l - 32; i >= 16; i -= 16) {
        AES_AddRoundKey(block, &key[i]);
        AES_MixColumns_Inv(block, AES_xtime);
        AES_ShiftRows(block, AES_ShiftRowTab_Inv);
        AES_SubBytes(block, AES_Sbox_Inv_init);
    }
    AES_AddRoundKey(block, &key[0]);

    for (int i = 0; i < 16; i++) {
        aes_block_array[0].block[i] = block[i];
    }
}

int dev() {

    //Вводимая строка
    char* inputLine = "Lorem ipsum dolor sit amet consectetur adipisicing elit. Minus repellat debitis possimus, ipsa doloribus quos ipsum, laboriosam quia at sapiente culpa iusto enim, voluptatem deserunt dignissimos! Ipsa sit rerum, totam.";
    std::cout << "Input line: " << inputLine << std::endl;

    //Ключ
    char* keyLine = "0123456789abcdef"; 
    std::cout << "Key: " << keyLine << std::endl;

    //Формируем key и keyLen
    BYTE key[16 * (14 + 1)];
    int keyLen = 0;
    for (int i = 0; keyLine[i] != '\0'; ++i) {
        key[keyLen++] = keyLine[i];
    }

    int expandKeyLen = AES_ExpandKey(key, keyLen, AES_Sbox_init);

    // Изучаем размер строки
    int fileLength = strlen(inputLine);
    int block_number = fileLength / 16;
    int incomplete_block_length = fileLength % 16;
    AES_block* aes_block_array;

    std::cout << "Incomplete block length: " << incomplete_block_length << std::endl;

    // Выделяем память под массив блоков данных
    if (incomplete_block_length != 0)
        aes_block_array = new AES_block[block_number + 1];
    else
        aes_block_array = new AES_block[block_number];
    char temp[16];

    // Считываем блоки данных из строки
    for (int i = 0; i < block_number; i++) {
        memcpy(temp, inputLine + i * 16, 16);
        for (int j = 0; j < 16; j++) {
            aes_block_array[i].block[j] = (unsigned char)temp[j];
        }
    }

    // Если есть остаточные байты, добавляем нули до полного блока
    if (incomplete_block_length != 0) {
        memcpy(temp, inputLine + block_number * 16, incomplete_block_length);
        for (int j = 0; j < 16; j++) {
            aes_block_array[block_number].block[j] = (unsigned char)temp[j];
        }
        for (int j = 1; j <= 16 - incomplete_block_length; j++)
            aes_block_array[block_number].block[16 - j] = '\0';
        block_number++;
    }

    AES_Encrypt(aes_block_array, key, expandKeyLen, block_number);

    std::cout << "Encrypted: " << std::endl;

    for (int i = 0; i < block_number - 1; i++) {
        printBytes(aes_block_array[i].block, 16);
    }
    printBytes(aes_block_array[block_number - 1].block, incomplete_block_length);
    
    AES_Decrypt(aes_block_array, key, expandKeyLen, block_number);

    std::cout << "Decrypted: " << std::endl;

    for (int i = 0; i < block_number - 1; i++) {
        printChars(aes_block_array[i].block, 16);
    }
    printChars(aes_block_array[block_number - 1].block, incomplete_block_length);

    return 0;

}